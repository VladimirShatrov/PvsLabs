#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <chrono>
#include <random>
#include <hip/hip_runtime.h>

std::vector<float> generate_random_array(size_t size) {
    std::vector<float> array(size);
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);

    for (size_t i = 0; i < size; ++i) {
        array[i] = dist(gen);
    }
    return array;
}

__global__ void sum_kernel_reduction(const float* array, float* result, int size) {
    extern __shared__ float shared_data[];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    shared_data[tid] = (idx < size) ? array[idx] : 0.0f;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(result, shared_data[0]);
    }
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << msg << " - " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
}

float parallel_sum(const std::vector<float>& array, int block_size) {
    float* d_array = nullptr;
    float* d_result = nullptr;
    float h_result = 0.0f;

    checkCudaError(hipMalloc(&d_array, array.size() * sizeof(float)), "hipMalloc d_array");
    checkCudaError(hipMalloc(&d_result, sizeof(float)), "hipMalloc d_result");

    checkCudaError(hipMemcpy(d_array, array.data(), array.size() * sizeof(float), hipMemcpyHostToDevice),
        "hipMemcpy H2D array");
    checkCudaError(hipMemcpy(d_result, &h_result, sizeof(float), hipMemcpyHostToDevice),
        "hipMemcpy H2D result");

    int grid_size = (array.size() + block_size - 1) / block_size;
    sum_kernel_reduction << <grid_size, block_size, block_size * sizeof(float) >> > (d_array, d_result, array.size());

    checkCudaError(hipGetLastError(), "Kernel execution");
    checkCudaError(hipDeviceSynchronize(), "Device sync");

    checkCudaError(hipMemcpy(&h_result, d_result, sizeof(float), hipMemcpyDeviceToHost),
        "hipMemcpy D2H result");

    hipFree(d_array);
    hipFree(d_result);

    return h_result;
}

int main(int argc, char** argv) {
    const size_t array_size = 10000000;
    const int num_runs = 100;
    const int block_size = 1024; // Оптимальный размер блока для большинства GPU
    double total_time = 0.0;
    float reference_sum = 0.0f;

    auto array = generate_random_array(array_size);

    // Первый запуск (без учета в среднее)
    auto start = std::chrono::high_resolution_clock::now();
    reference_sum = parallel_sum(array, block_size);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;

    std::cout << "Reference sum: " << reference_sum << std::endl;
    std::cout << "First run time: " << duration.count() << " seconds\n\n";

    // Основные запуски (100 раз)
    for (int i = 0; i < num_runs; ++i) {
        start = std::chrono::high_resolution_clock::now();
        float sum = parallel_sum(array, block_size);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        total_time += duration.count();

    }

    // Вывод результатов
    double avg_time = total_time / num_runs;
    std::cout << "Block size: " << block_size << std::endl;
    std::cout << "Average time over " << num_runs << " runs: " << avg_time << " seconds" << std::endl;
    std::cout << "Total time: " << total_time << " seconds" << std::endl;

    return 0;
}